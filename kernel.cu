#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    __device__ __shared__ float partialsum[BLOCK_SIZE * 2];
    unsigned int t = threadIdx.x;
    unsigned int start= blockIdx.x*blockDim.x *2;
    if(t+start<size)
    {
    partialsum[t]=in[start+t];
    }
    else
    partialsum[t]=0.0;
    if((blockDim.x+t+start)<size)
    {
    partialsum[blockDim.x+t]=in[start+blockDim.x+t]; 
    }
    else
    partialsum[blockDim.x+t]=0.0;
    //__syncthreads();
    for(unsigned int stride=blockDim.x;stride>=1;stride/=2)
    {
    __syncthreads();
    if(t<stride)
    {
    partialsum[t]+=partialsum[t+stride]; 
    }
    }
    out[blockIdx.x]= partialsum[0];
}
