#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    __device__ __shared__ float partialsum[BLOCK_SIZE * 2];
    unsigned int t = threadIdx.x;
    unsigned int start= blockIdx.x*blockDim.x *2;
    if(t<size)
    {
    partialsum[t]=in[start+t];
    }
    if((blockDim.x+t)<size)
    {
    partialsum[blockDim.x+t]=in[start+blockDim.x+t]; 
    }
    __syncthreads();
    for(unsigned int stride=blockDim.x;stride>=1;stride/=2)
    {
    //__syncthreads();
    if(t<stride)
    {
    partialsum[t]+=partialsum[t+stride]; 
    }
    }
    out[blockIdx.x]= partialsum[0];
}
